#include "hip/hip_runtime.h"
#include "ParticleRenderer.hpp"
#include "ScopedTimer.hpp"
#include "Grid3.hpp"
#include <iostream>
#include <cmath>
#include <string>
#include <algorithm>
#include <functional>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/gtx/norm.hpp>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#ifndef M_PI
#define M_PI 3.14159
#endif

template <int N> constexpr float pow(float x) { return x*pow<N-1>(x); }
template <> constexpr float pow<1>(float x) { return x; }
constexpr float sq(float x) { return x*x; }

struct Particle {
    int nextParticle;
    glm::vec3 position;
    glm::vec3 lastPosition;
    glm::vec3 velocity;
    glm::vec3 lastVelocity;
    glm::vec3 accel;
    float time;
    float density;
    float lastDensity;
    float pressure;
    float lastPressure;


    __host__ __device__
    Particle(const Particle& copy) = default;

    __host__ __device__
    Particle(glm::vec3 position)
        : nextParticle(-1)
        , position(position)
        , lastPosition(position)
        , velocity(0.0f,0.0f,0.0f)
        , lastVelocity(velocity)
        , accel(0.0f,0.0f,0.0f)
        , time(0.0f)
        , density(rho0)
        , lastDensity(rho0)
        , pressure(0)
        , lastPressure(0)
        {}
    __host__ __device__
    Particle()
        : Particle(glm::vec3(0.0f,0.0f,0.0f))
        {}

    static constexpr float mass = 0.524f;
    static constexpr float radius = 0.05f;
    static constexpr float cs = 1400.0f; // m/s^2
    static constexpr float rho0 = 1000; // water density
    static constexpr int gamma = 7;
};

__device__
Particle backtrace(const Particle& particle, float t)
{
    Particle res(particle);
    res.position = glm::mix(particle.lastPosition, particle.position, t);
    res.velocity = glm::mix(particle.lastVelocity, particle.velocity, t);
    res.density = glm::mix(particle.lastDensity, particle.density, t);
    res.pressure = glm::mix(particle.lastPressure, particle.pressure, t);
    return res;
}

__device__
float W(glm::vec3 disp) {
    float r = length(disp);
    if (r > Particle::radius)
        return 0.0f;
    float x = 1.0f - r / Particle::radius;
    return 315.0f / (64.0f * M_PI * pow<3>(Particle::radius)) * pow<3>(x);
}

__device__
glm::vec3 dW(glm::vec3 disp) {
    float r = length(disp);
    if (r > Particle::radius)
        return glm::vec3(0.0f,0.0f,0.0f);
    if (r > 0.0f) disp /= r;
    float x = 1.0f - r / Particle::radius;
    return disp * float(-45.0f / (M_PI * pow<4>(Particle::radius)) * sq(x));
}

struct Cell {
    int firstParticle;
    float time;
    float deltaTime;
    int updateCounter;

    Cell()
        : firstParticle(-1)
        , time(0.0f)
        , deltaTime(std::numeric_limits<float>::max()) {}
};

std::vector<Particle> particles;
std::vector<Cell> cells(200*200*200);

void updateCells() {
    for (int i = 0; i < cells.size(); i++) {
        cells[i].firstParticle = -1;
        cells[i].deltaTime = 0.01f;//std::numeric_limits<float>::max();
        cells[i].time = std::numeric_limits<float>::max();
        cells[i].updateCounter = 0;
    }
    
    for (int i = 0; i < particles.size(); i++) {
        glm::ivec3 index = particles[i].position / Particle::radius;
        index = glm::min(glm::max(index, glm::ivec3(0,0,0)), glm::ivec3(200,200,200)-1);
        int k = index.z*200*200+index.y*200+index.x;
        particles[i].nextParticle = cells[k].firstParticle;
        cells[k].firstParticle = i;

        float dt = min(
            0.02f*sqrt(Particle::radius/(0.001f+length(particles[i].accel))),
            0.05f*Particle::radius/(0.001f+length(particles[i].velocity))
        );
        cells[k].deltaTime = std::min(dt, cells[k].deltaTime);
        cells[k].time = std::min(particles[i].time, cells[k].time);
    }
}

__device__
Particle stepParticle(Particle& particle_, Particle* neighbors, int numNeighbors, float dt) {
    Particle particle = particle_;
    particle.lastPosition = particle.position;
    particle.lastVelocity = particle.velocity;
    particle.lastDensity = particle.density;
    particle.lastPressure = particle.pressure;
    
    // Compute density
    particle.density = 0.0f;
    for (int i = 0; i < numNeighbors; i++) {
        auto xij = particle.position - neighbors[i].position;
        particle.density += Particle::mass*W(xij);
    }

    // Compute F* (Fvisc + Fext)
    constexpr float nu = 0.015f;
    particle.accel = glm::vec3(0.0f, -9.81f, 0.0f);
    for (int i = 0; i < numNeighbors; i++) {
        auto vij = particle.velocity - neighbors[i].velocity;
        auto xij = particle.position - neighbors[i].position;
        if (length2(xij) > 0.0) {
            particle.accel += 2.0f * nu * Particle::mass / neighbors[i].density
                * vij * dot(xij, dW(xij))
                / (dot(xij, xij) + 0.01f * sq(Particle::radius));
        }
    }

    // Compute velocity using forces
    particle.velocity += dt*particle.accel;
    
    // Compute new density
    particle.density = 0.0f;
    for (int i = 0; i < numNeighbors; i++) {
        auto xij = particle.position - neighbors[i].position;
        auto vij = particle.velocity - neighbors[i].velocity;
        particle.density += Particle::mass*W(xij);
        particle.density += dt*dot(dW(xij), vij);
    }

    // Compute pressure and pressure forces
    constexpr float kappa = 0.5f;
    auto accelP = glm::vec3{0.0f,0.0f,0.0f};
    particle.pressure = kappa*std::max(particle.density - Particle::rho0, 0.0f);
    for (int i = 0; i < numNeighbors; i++) {
        auto xij = particle.position - neighbors[i].position;
        if (length2(xij) > 0.0) {
            accelP -= dW(xij) * Particle::mass
                * (particle.pressure / sq(particle.density)
                + neighbors[i].pressure / sq(neighbors[i].density));
        }
    }
    particle.accel += accelP;

    // Integrate particle over time using dt
    particle.velocity += dt*accelP;
    particle.position += dt*particle.velocity + sq(dt)/2.0f*particle.accel;

    glm::vec3 r = glm::vec3(0.4f,0.3f,0.4f);
    glm::vec3 a = 5.0f-r, b = 5.0f+r;
    for (int d = 0; d < 3; d++) {
        if (particle.position[d] < a[d]) {
            particle.position[d] = a[d]+std::min(b[d]-a[d], a[d]-particle.position[d]);
            particle.velocity[d] *= -0.2f;
        }
        if (particle.position[d] > b[d]) {
            particle.position[d] = b[d]-std::min(b[d]-a[d], particle.position[d]-b[d]);
            particle.velocity[d] *= -0.2f;
        }
    }
    particle.time += dt;
    return particle;
}

__global__
void stepCell(Cell* cells, Cell* nextCells, Particle* particles, Particle* nextParticles, float timeLimit) {
    constexpr static glm::ivec3 deltas[] = {
        glm::ivec3(-1,-1,-1),
        glm::ivec3(-1,-1, 0),
        glm::ivec3(-1,-1,+1),
        glm::ivec3(-1, 0,-1),
        glm::ivec3(-1, 0, 0),
        glm::ivec3(-1, 0,+1),
        glm::ivec3(-1,+1,-1),
        glm::ivec3(-1,+1, 0),
        glm::ivec3(-1,+1,+1),
        glm::ivec3( 0,-1,-1),
        glm::ivec3( 0,-1, 0),
        glm::ivec3( 0,-1,+1),
        glm::ivec3( 0, 0,-1),
        glm::ivec3( 0, 0,+1),
        glm::ivec3( 0,+1,-1),
        glm::ivec3( 0,+1, 0),
        glm::ivec3( 0,+1,+1),
        glm::ivec3(+1,-1,-1),
        glm::ivec3(+1,-1, 0),
        glm::ivec3(+1,-1,+1),
        glm::ivec3(+1, 0,-1),
        glm::ivec3(+1, 0, 0),
        glm::ivec3(+1, 0,+1),
        glm::ivec3(+1,+1,-1),
        glm::ivec3(+1,+1, 0),
        glm::ivec3(+1,+1,+1),
    };

    glm::ivec3 center(90+blockIdx.x, 90+blockIdx.y, 90+blockIdx.z);
    int centerIndex = center.x+center.y*200+center.z*200*200;

    nextCells[centerIndex] = cells[centerIndex];
    nextCells[centerIndex].updateCounter = 0;
    
    Cell& centerCell = cells[centerIndex];
    if (centerCell.firstParticle == -1 || centerCell.time > timeLimit)
        return;
    
    __shared__ Particle neighbors[160];
    __shared__ int numNeighbors;
    __shared__ float neighborTime;
    __shared__ bool doStep;

    if (threadIdx.x == 0) {
        doStep = true;
        numNeighbors = 0;

        for (int i = centerCell.firstParticle; i != -1; i = particles[i].nextParticle) {
            neighbors[numNeighbors++] = particles[i];
            assert(numNeighbors != 160);
        }

        neighborTime = std::numeric_limits<float>::max();
        for (glm::ivec3 delta : deltas) {
            const glm::ivec3 index = center+delta;
            const Cell& cell = cells[index.z*200*200+index.y*200+index.x];
            if (cell.firstParticle == -1)
                continue;
            if (cell.time < centerCell.time) {
                doStep = false;
            }
            neighborTime = min(neighborTime, cell.time);
            for (int i = cell.firstParticle; i != -1; i = particles[i].nextParticle) {
                float t = (centerCell.time - cell.time + cell.deltaTime) / cell.deltaTime;
                neighbors[numNeighbors++] = backtrace(particles[i], t);
                assert(numNeighbors != 160);
            }
        }
    }
Sync:
    __syncthreads();
   

    for (int k = 0, i = centerCell.firstParticle; i != -1; i = particles[i].nextParticle, k++) {
        if (k == threadIdx.x) {
            if (doStep)
                nextParticles[i] = stepParticle(neighbors[k], neighbors, numNeighbors, centerCell.deltaTime);
            else
                nextParticles[i] = particles[i];
            break;
        }
    }

    if (threadIdx.x == 0 && doStep) {
        nextCells[centerIndex].updateCounter = 1;
        nextCells[centerIndex].time = centerCell.time+centerCell.deltaTime;
    }
}

void backtrackAll(std::vector<glm::vec3>& result, float time)
{
    result.clear();
    for (int i = 0; i < particles.size(); i++) {
        glm::ivec3 index = particles[i].position / Particle::radius;
        if (index.x < 0)    index.x = 0;
        if (index.x >= 200) index.x = 199;
        if (index.y < 0)    index.y = 0;
        if (index.y >= 200) index.y = 199;
        if (index.z < 0)    index.z = 0;
        if (index.z >= 200) index.z = 199;
        int k = index.x+index.y*200+index.z*200*200;
        float t = (time - cells[k].time + cells[k].deltaTime) / (cells[k].deltaTime);
        result.push_back(mix(particles[i].lastPosition, particles[i].position, t));
    }
}

void packSphere(const glm::vec3& center, float radius) {
    int r = (2.0f*radius) / Particle::radius;
    for (int z = -r; z <= r; z++)
    for (int y = -r; y <= r; y++)
    for (int x = -r; x <= r; x++) {
        auto pos = Particle::radius*(
            glm::vec3(0.0f,0.5f,0.5f)*float(x)
          + glm::vec3(0.5f,0.0f,0.5f)*float(y)
          + glm::vec3(0.5f,0.5f,0.0f)*float(z));
        if (length2(pos) < sq(radius)) {
            particles.emplace_back(center+pos);
        }
    }
}

struct isUpdated {
    __device__ __host__
    bool operator()(Cell cell) {
        return cell.updateCounter;
    }
};


__device__
bool operator <(Particle a, Particle b) {
    glm::ivec3 indexA = a.position/Particle::radius;
    glm::ivec3 indexB = b.position/Particle::radius;
    return (indexA.z-indexB.z)*200*200
            +(indexA.y-indexB.y)*200
            +(indexA.x-indexB.x) < 0;
}

/*__global__
void update(Cell* cells, Cell* nextCells, Particle* particles, Particle* nextParticles, int numParticles, float time) {
    for (int k = 0; k < 50; k++) {
        __syncthreads();
        stepCell<<<dim3(20,20,20),1>>>(cells, nextCells, particles, nextParticles, time);
        hipDeviceSynchronize();
        __syncthreads();
        memcpy(cells, nextCells, 200*200*200*sizeof(Cell));
        memcpy(particles, nextParticles, numParticles*sizeof(Particle));
    }
}*/

int main(int argc, char **argv)
{
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <duration> <path>" << std::endl;
        return EXIT_FAILURE;
    }
    const float duration = std::stof(argv[1]);
    const std::string path = argv[2];

    ParticleRenderer renderer(512,512);
    packSphere({5.0f,5.0f,5.0f}, 0.25f);

    int numParticles = particles.size();
    Particle *particles_dev;
    Particle *nextParticles_dev;
    Cell *cells_dev;
    Cell *nextCells_dev;
    hipMalloc(&particles_dev, numParticles*sizeof(Particle));
    hipMalloc(&nextParticles_dev, numParticles*sizeof(Particle));
    hipMalloc(&cells_dev, 200*200*200*sizeof(Cell));
    hipMalloc(&nextCells_dev, 200*200*200*sizeof(Cell));
    hipMemcpy(particles_dev, particles.data(), numParticles*sizeof(Particle), hipMemcpyHostToDevice);
    hipMemcpy(nextParticles_dev, particles_dev, numParticles*sizeof(Particle), hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();

    float time = 0.0f;
    int frameCount = 0;
    std::string filename = path+"XXXX.png";
    std::vector<glm::vec3> positions;
    while (time <= duration) {
        time += 1.0f/60.0f;
        
        ScopedTimer timer(std::to_string(frameCount));
        updateCells();
        hipMemcpyAsync(particles_dev, particles.data(), numParticles*sizeof(Particle), hipMemcpyHostToDevice);
        hipMemcpyAsync(nextParticles_dev, particles_dev, numParticles*sizeof(Particle), hipMemcpyDeviceToDevice);
        hipMemcpyAsync(cells_dev, cells.data(), 200*200*200*sizeof(Cell), hipMemcpyHostToDevice);
        hipMemcpyAsync(nextCells_dev, cells_dev, 200*200*200*sizeof(Cell), hipMemcpyDeviceToDevice);
        //step<<<1,1>>>(res_dev, cells_dev, cellsNext_dev, particles_dev, particlesNext_dev, numParticles, time);
        
        for (;;) {
            for (int k = 0; k < 50; k++) {
                stepCell<<<dim3(20,20,20),160>>>(cells_dev, nextCells_dev, particles_dev, nextParticles_dev, time);
                //hipMemcpyAsync(cells_dev, nextCells_dev, 200*200*200*sizeof(Cell), hipMemcpyDeviceToDevice);
                std::swap(cells_dev, nextCells_dev);
                //hipMemcpyAsync(particles_dev, nextParticles_dev, numParticles*sizeof(Particle), hipMemcpyDeviceToDevice);
                std::swap(particles_dev, nextParticles_dev);
            }
            //update<<<1,1>>>(cells_dev, nextCells_dev, particles_dev, nextParticles_dev, numParticles, time);
            hipDeviceSynchronize();

            thrust::device_ptr<Cell> p = thrust::device_pointer_cast(cells_dev);
            if (thrust::find_if(thrust::device, p, p+200*200*200, isUpdated()) == p+200*200*200)
                break;
        }
        /*{
            thrust::device_ptr<Particle> p = thrust::device_pointer_cast(particles_dev);
            thrust::sort(thrust::device, p, p+numParticles);
        }*/

        hipMemcpyAsync(particles.data(), particles_dev, numParticles*sizeof(Particle), hipMemcpyDeviceToHost);
        hipMemcpyAsync(cells.data(), cells_dev, 200*200*200*sizeof(Cell), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();


        backtrackAll(positions, time);
        renderer.render(positions);

        int rem = frameCount;
        for (auto p = filename.rbegin()+4; p != filename.rbegin()+8; ++p) {
            *p = '0' + (rem % 10);
            rem /= 10;
        }
        //std::cerr << "writing " << filename << " at " << time << "s" << std::endl;

        stbi_flip_vertically_on_write(1);
        stbi_write_png(filename.c_str(), renderer.width(), renderer.height(),
            3, renderer.frontBuffer().data(), 0);
        
        frameCount++;
    }
    //std::cerr << frameCount << " frames total." << std::endl;
    return EXIT_SUCCESS;
}

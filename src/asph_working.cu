#include "hip/hip_runtime.h"
#include "ParticleRenderer.hpp"
#include "ScopedTimer.hpp"
#include "Grid3.hpp"
#include <iostream>
#include <cmath>
#include <string>
#include <algorithm>
#include <functional>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/gtx/norm.hpp>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

template <int N> constexpr float pow(float x) { return x*pow<N-1>(x); }
template <> constexpr float pow<1>(float x) { return x; }
constexpr float sq(float x) { return x*x; }

struct Particle {
    float time;
    float deltaTime;
    int nextParticle;
    glm::vec3 position;
    glm::vec3 lastPosition;
    glm::vec3 velocity;
    glm::vec3 lastVelocity;
    glm::vec3 accel;
    float density;
    float lastDensity;
    float pressure;
    float lastPressure;


    __host__ __device__
    Particle(const Particle& copy) = default;
    __host__ __device__
    Particle(glm::vec3 position)
        : time(0.0f)
        , deltaTime(0.0f)
        , nextParticle(-1)
        , position(position)
        , lastPosition(position)
        , velocity(0.0f,0.0f,0.0f)
        , lastVelocity(velocity)
        , accel(0.0f,0.0f,0.0f)
        , density(rho0)
        , lastDensity(rho0)
        , pressure(0)
        , lastPressure(0)
        {}
    __host__ __device__
    Particle()
        : Particle(glm::vec3(0.0f,0.0f,0.0f))
        {}

    static constexpr float mass = 0.524f;
    static constexpr float radius = 0.05f;
    static constexpr float cs = 1400.0f; // m/s^2
    static constexpr float rho0 = 1000; // water density
    static constexpr int gamma = 7;
};

__host__ __device__
Particle backtrace(const Particle& particle, float t)
{
    Particle res(particle);
    res.position = glm::mix(particle.lastPosition, particle.position, t);
    res.velocity = glm::mix(particle.lastVelocity, particle.velocity, t);
    res.density = glm::mix(particle.lastDensity, particle.density, t);
    res.pressure = glm::mix(particle.lastPressure, particle.pressure, t);
    return res;
}

__host__ __device__
float W(glm::vec3 disp) {
    float r = length(disp);
    float x = 1.0f - r / Particle::radius;
    return 315.0f / (64.0f * M_PI * pow<3>(Particle::radius)) * pow<3>(x);
}

__host__ __device__
glm::vec3 dW(glm::vec3 disp) {
    float r = length(disp);
    if (r > 0.0f) disp /= r;
    float x = 1.0f - r / Particle::radius;
    return disp * float(-45.0f / (M_PI * pow<4>(Particle::radius)) * sq(x));
}

void updateCells(std::vector<int>& cells, std::vector<Particle>& particles) {
    std::fill(cells.begin(),cells.end(),-1);
    for (int i = 0; i < particles.size(); i++) {
        glm::ivec3 index = particles[i].position / Particle::radius;
        index = glm::min(glm::max(index, glm::ivec3(0,0,0)), glm::ivec3(200,200,200)-1);
        int k = index.z*200*200+index.y*200+index.x;
        particles[i].nextParticle = cells[k];
        cells[k] = i;
    }
}

__global__
void step(Particle* particles, Particle* particlesNext, int* cells) {
    int k = blockIdx.x;

    // Reconstruct neighbor attributes
    Particle neighbors[128];
    int numNeighbors = 0;

    Particle& particle = neighbors[numNeighbors++] = particles[k];
    particle.lastPosition = particle.position;
    particle.lastVelocity = particle.velocity;
    particle.lastDensity = particle.density;
    particle.lastPressure = particle.pressure;

    // Determine possible time step dt
    float dt = min(
        0.02f*sqrt(Particle::radius/(0.001f+length(particles[k].accel))),
        0.05f*Particle::radius/(0.001f+length(particles[k].velocity))
    );
    particle.deltaTime = dt;

    glm::ivec3 center = particles[k].position / Particle::radius;
    for (int dk = -1; dk <= 1; dk++)
    for (int dj = -1; dj <= 1; dj++)
    for (int di = -1; di <= 1; di++) {
        glm::ivec3 index = center+glm::ivec3(di,dj,dk);
        for (int i = cells[index.z*200*200+index.y*200+index.x];
                i != -1;
                i = particles[i].nextParticle) {
            Particle& neighbor = particles[i];
            if (k == i)
                continue;
            if (neighbor.time < particle.time)
                return;

            float t = 0.0f;
            if (neighbor.deltaTime != 0)
                t = (neighbors[0].time - neighbor.time + neighbor.deltaTime)
                    / (neighbor.deltaTime);

            auto backtrack = backtrace(neighbor, t);
            if (distance2(backtrack.position, neighbors[0].position) < sq(Particle::radius)) {
                if (numNeighbors == 128) {
                    printf("error!\n");
                    return;
                }
                neighbors[numNeighbors++] = backtrack;
            }
        }
    }    

    // Compute density
    particle.density = 0.0f;
    for (int i = 0; i < numNeighbors; i++) {
        auto xij = particle.position - neighbors[i].position;
        particle.density += Particle::mass*W(xij);
    }

    // Compute F* (Fvisc + Fext)
    constexpr float nu = 0.015f;
    particle.accel = glm::vec3(0.0f, -9.81f, 0.0f);
    for (int i = 0; i < numNeighbors; i++) {
        auto vij = particle.velocity - neighbors[i].velocity;
        auto xij = particle.position - neighbors[i].position;
        if (length2(xij) > 0.0) {
            particle.accel += 2.0f * nu * Particle::mass / neighbors[i].density
                * vij * dot(xij, dW(xij))
                / (dot(xij, xij) + 0.01f * sq(Particle::radius));
        }
    }

    // Compute velocity using forces
    particle.velocity += dt*particle.accel;
    
    // Compute new density
    particle.density = 0.0f;
    for (int i = 0; i < numNeighbors; i++) {
        auto xij = particle.position - neighbors[i].position;
        auto vij = particle.velocity - neighbors[i].velocity;
        particle.density += Particle::mass*W(xij);
        particle.density += dt*dot(dW(xij), vij);
    }

    // Compute pressure and pressure forces
    constexpr float kappa = 0.5f;
    auto accelP = glm::vec3{0.0f,0.0f,0.0f};
    particle.pressure = kappa*std::max(particle.density - Particle::rho0, 0.0f);
    for (int i = 0; i < numNeighbors; i++) {
        auto xij = particle.position - neighbors[i].position;
        if (length2(xij) > 0.0) {
            accelP -= dW(xij) * Particle::mass
                * (particle.pressure / sq(particle.density)
                + neighbors[i].pressure / sq(neighbors[i].density));
        }
    }
    particle.accel += accelP;

    // Integrate particle over time using dt
    particle.velocity += dt*accelP;
    particle.position += dt*particle.velocity + sq(dt)/2.0f*particle.accel;
    particle.time += dt;

    glm::vec3 r = glm::vec3(0.4f,0.3f,0.4f);
    glm::vec3 a = 5.0f-r, b = 5.0f+r;
    for (int d = 0; d < 3; d++) {
        if (particle.position[d] < a[d]) {
            particle.position[d] = a[d]+std::min(b[d]-a[d], a[d]-particle.position[d]);
            particle.velocity[d] *= -0.2f;
        }
        if (particle.position[d] > b[d]) {
            particle.position[d] = b[d]-std::min(b[d]-a[d], particle.position[d]-b[d]);
            particle.velocity[d] *= -0.2f;
        }
    }

    particlesNext[k] = particle;
}

void backtrackAll(std::vector<glm::vec3>& result, std::vector<Particle>& particles, float time)
{
    result.clear();
    for (int i = 0; i < particles.size(); i++) {
        float t = 0.0f;
        if (particles[i].deltaTime != 0)
            t = (time - particles[i].time + particles[i].deltaTime)
                / (particles[i].deltaTime);
        result.push_back(mix(particles[i].lastPosition, particles[i].position, t));
    }
}

void packSphere(std::vector<Particle>& particles, const glm::vec3& center, float radius) {
    int r = (2.0f*radius) / Particle::radius;
    for (int z = -r; z <= r; z++)
    for (int y = -r; y <= r; y++)
    for (int x = -r; x <= r; x++) {
        auto pos = Particle::radius*(
            glm::vec3(0.0f,0.5f,0.5f)*float(x)
          + glm::vec3(0.5f,0.0f,0.5f)*float(y)
          + glm::vec3(0.5f,0.5f,0.0f)*float(z));
        if (length2(pos) < sq(radius)) {
            particles.emplace_back(center+pos);
        }
    }
}


int main(int argc, char **argv)
{
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <duration> <path>" << std::endl;
        return EXIT_FAILURE;
    }
    const float duration = std::stof(argv[1]);
    const std::string path = argv[2];

    ParticleRenderer renderer(512,512);
    std::vector<Particle> particles;
    packSphere(particles, {5.0f,5.0f,5.0f}, 0.25f);
    std::vector<int> cells(200*200*200);

    int numParticles = particles.size();
    Particle *particles_dev;
    Particle *particlesNext_dev;
    int *cells_dev;
    hipMalloc(&particles_dev, numParticles*sizeof(Particle));
    hipMalloc(&particlesNext_dev, numParticles*sizeof(Particle));
    hipMalloc(&cells_dev, 200*200*200*sizeof(int));
    hipMemcpy(particles_dev, particles.data(), numParticles*sizeof(Particle), hipMemcpyHostToDevice);
    hipMemcpy(particlesNext_dev, particles_dev, numParticles*sizeof(Particle), hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();

    float time = 0.0f;
    int frameCount = 0;
    std::string filename = path+"XXXX.png";
    std::vector<glm::vec3> positions;
    while (time <= duration) {
        time += 1.0f/60.0f;
        float minTime;
        {
            ScopedTimer timer(std::to_string(frameCount));
            updateCells(cells, particles);
            hipMemcpy(particles_dev, particles.data(), numParticles*sizeof(Particle), hipMemcpyHostToDevice);
            hipMemcpy(cells_dev, cells.data(), 200*200*200*sizeof(int), hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            hipMemcpyAsync(particlesNext_dev, particles_dev, numParticles*sizeof(Particle), hipMemcpyDeviceToDevice);
            
            float totalMilliseconds = 0.0f;
            do {
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);

                hipEventRecord(start);
                step<<<numParticles,1>>>(particles_dev, particlesNext_dev, cells_dev);
                hipMemcpy(particles.data(), particlesNext_dev, numParticles*sizeof(Particle), hipMemcpyDeviceToHost);
                hipEventRecord(stop);
                hipDeviceSynchronize();
                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                totalMilliseconds += milliseconds;

                minTime = particles.front().time;
                for (int i = 1; i < numParticles; i++) {
                    glm::ivec3 index = particles[i].position/Particle::radius;
                    if (minTime > particles[i].time) {
                        minTime = particles[i].time;
                    }
                }
                hipMemcpyAsync(particles_dev, particlesNext_dev, numParticles*sizeof(Particle), hipMemcpyDeviceToDevice);
            } while (minTime < time);

            std::cout << "kernel time total " << totalMilliseconds << std::endl;
        }
        backtrackAll(positions, particles, time);
        renderer.render(positions);

        int rem = frameCount;
        for (auto p = filename.rbegin()+4; p != filename.rbegin()+8; ++p) {
            *p = '0' + (rem % 10);
            rem /= 10;
        }
        std::cerr << "writing " << filename << " at " << time << "s" << std::endl;

        stbi_flip_vertically_on_write(1);
        stbi_write_png(filename.c_str(), renderer.width(), renderer.height(),
            3, renderer.frontBuffer().data(), 0);
        
        frameCount++;
    }
    std::cerr << frameCount << " frames total." << std::endl;
    return EXIT_SUCCESS;
}
